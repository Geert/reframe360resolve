#include "hip/hip_runtime.h"
#include "hip/hip_vector_types.h"

#define PI 3.1415926535897932384626433832795

__device__ float3 matMul(const float3 r012, const float3 r345, const float3 r678, float3 v){
	float3 outvec = { 0, 0, 0 };
	outvec.x = r012.x * v.x + r012.y * v.y + r012.z * v.z;
	outvec.y = r345.x * v.x + r345.y * v.y + r345.z * v.z;
	outvec.z = r678.x * v.x + r678.y * v.y + r678.z * v.z;
	return outvec;
}

__device__ float2 repairUv(float2 uv){
	float2 outuv = {0, 0};

	if(uv.x<0) {
		outuv.x = 1.0 + uv.x;
		}else if(uv.x > 1.0){
			outuv.x = uv.x -1.0;
		} else {
			outuv.x = uv.x;
		}

		if(uv.y<0) {
			outuv.y = 1.0 + uv.y;
		} else if(uv.y > 1.0){
			outuv.y = uv.y -1.0;
		} else {
			outuv.y = uv.y;
		}

	return outuv;
}

__device__ float2 polarCoord(float3 dir) {	
	float3 ndir = normalize(dir);
	float longi = -atan2(ndir.z, ndir.x);
	
	float lat = acos(-ndir.y);
	
	float2 uv;
	uv.x = longi;
	uv.y = lat;
	
	float2 pitwo = {PI, PI};
	uv /= pitwo;
	uv.x /= 2.0;
	float2 ones = {1.0, 1.0};
	uv = fmodf(uv, ones);
	return uv;
}


__device__ float3 fisheyeDir(float3 dir, const float3 r012, const float3 r345, const float3 r678) {
	
	dir.x = dir.x / dir.z;
	dir.y = dir.y / dir.z;
	dir.z = dir.z / dir.z;
	
	float2 uv;
	uv.x = dir.x;
	uv.y = dir.y;
	float r = sqrtf(uv.x*uv.x + uv.y*uv.y);
	
	float phi = atan2f(uv.y, uv.x);
	
	float theta = r;
	
	float3 fedir = { 0, 0, 0 };
	fedir.x = sin(theta) * cos(phi);
	fedir.y = sin(theta) * sin(phi);
	fedir.z = cos(theta);

	fedir = matMul(r012, r345, r678, fedir);
	
	return fedir;
}

__device__ float4 linInterpCol(float2 uv, const float* input, int width, int height){
	float4 outCol = {0,0,0,0};
	float i = floor(uv.x);
	float j = floor(uv.y);
	float a = uv.x-i;
	float b = uv.y-j;
	int x = (int)i;
	int y = (int)j;
	const int indexX1Y1 = ((y * width) + x) * 4;
	const int indexX2Y1 = ((y * width) + x+1) * 4;
	const int indexX1Y2 = (((y+1) * width) + x) * 4;
	const int indexX2Y2 = (((y+1) * width) + x+1) * 4;
	const int maxIndex = (width * height -1) * 4;
	
	if(indexX2Y2 < maxIndex-height - 100){
		outCol.x = (1.0 - a)*(1.0 - b)*input[indexX1Y1] + a*(1.0 - b)*input[indexX2Y1] + (1.0 - a)*b*input[indexX1Y2] + a*b*input[indexX2Y2];
		outCol.y = (1.0 - a)*(1.0 - b)*input[indexX1Y1 + 1] + a*(1.0 - b)*input[indexX2Y1 + 1] + (1.0 - a)*b*input[indexX1Y2 + 1] + a*b*input[indexX2Y2 + 1];
		outCol.z = (1.0 - a)*(1.0 - b)*input[indexX1Y1 + 2] + a*(1.0 - b)*input[indexX2Y1 + 2] + (1.0 - a)*b*input[indexX1Y2 + 2] + a*b*input[indexX2Y2 + 2];
		outCol.w = (1.0 - a)*(1.0 - b)*input[indexX1Y1 + 3] + a*(1.0 - b)*input[indexX2Y1 + 3] + (1.0 - a)*b*input[indexX1Y2 + 3] + a*b*input[indexX2Y2 + 3];
	} else {
		outCol.x = input[indexX1Y1];
		outCol.y = input[indexX1Y1+ 1];
		outCol.z = input[indexX1Y1+ 2];
		outCol.w = input[indexX1Y1+ 3];
	}
	return outCol;
}

__global__ void GainAdjustKernel(int p_Width, int p_Height, float* p_Fov, float* p_Fisheye,
								const float* p_Input, float* p_Output, const float* r, int samples)
{
   const int x = blockIdx.x * blockDim.x + threadIdx.x;
   const int y = blockIdx.y * blockDim.y + threadIdx.y;

   if ((x < p_Width) && (y < p_Height))
   {
		const int index = ((y * p_Width) + x) * 4;

		float4 accum_col = {0, 0, 0, 0};

		for(int i=0; i<samples; i++){
			float fov = p_Fov[i];

		   float2 uv = { (float)x / p_Width, (float)y / p_Height };
		   float aspect = (float)p_Width / (float)p_Height;

		   float3 dir = { 0, 0, 0 };
		   dir.x = (uv.x - 0.5)*2.0;
		   dir.y = (uv.y - 0.5)*2.0;
		   dir.y /= aspect;
		   dir.z = fov;

		   const float3 r012 = {r[i*9+0], r[i*9+1], r[i*9+2]};
		   const float3 r345 = {r[i*9+3], r[i*9+4], r[i*9+5]};
		   const float3 r678 = {r[i*9+6], r[i*9+7], r[i*9+8]};

		   float3 rectdir = matMul(r012, r345, r678, dir);

		   rectdir = normalize(rectdir);

		   dir = lerp(rectdir, fisheyeDir(dir, r012, r345, r678), p_Fisheye[i]);

		   float2 iuv = polarCoord(dir);
		   iuv = repairUv(iuv);

		   int x_new = iuv.x * (p_Width - 1);
		   int y_new = iuv.y * (p_Height - 1);

		   iuv.x *= (p_Width - 1);
		   iuv.y *= (p_Height - 1);

		   if ((x_new < p_Width) && (y_new < p_Height))
		   {
			   const int index_new = ((y_new * p_Width) + x_new) * 4;

			   //float4 interpCol = linInterpCol(iuv, p_Input, p_Width, p_Height);
			   float4 interpCol = {p_Input[index_new + 0], p_Input[index_new + 1], p_Input[index_new + 2], p_Input[index_new + 3]};

			   accum_col.x += interpCol.x;
			   accum_col.y += interpCol.y;
			   accum_col.z += interpCol.z;
			   accum_col.w += interpCol.w;
			}
		}
		p_Output[index + 0] = accum_col.x / samples;
		p_Output[index + 1] = accum_col.y / samples;
		p_Output[index + 2] = accum_col.z / samples;
		p_Output[index + 3] = accum_col.w / samples;
   }
}

void RunCudaKernel(int p_Width, int p_Height, float* p_Fov, float* p_Fisheye, const float* p_Input, float* p_Output, const float* p_RotMat, int p_Samples)
{
    dim3 threads(128, 1, 1);
    dim3 blocks(((p_Width + threads.x - 1) / threads.x), p_Height, 1);

	float* dev_rmat;
	hipMalloc((void**)&dev_rmat, sizeof(float)*9*p_Samples);
	hipMemcpy((void*)dev_rmat, (void*)p_RotMat, sizeof(float)*9*p_Samples, hipMemcpyHostToDevice);

	float* dev_fov;
	hipMalloc((void**)&dev_fov, sizeof(float)*p_Samples);
	hipMemcpy((void*)dev_fov, (void*)p_Fov, sizeof(float)*p_Samples, hipMemcpyHostToDevice);

	float* dev_fisheye;
	hipMalloc((void**)&dev_fisheye, sizeof(float)*p_Samples);
	hipMemcpy((void*)dev_fisheye, (void*)p_Fisheye, sizeof(float)*p_Samples, hipMemcpyHostToDevice);

    GainAdjustKernel<<<blocks, threads>>>(p_Width, p_Height, dev_fov, dev_fisheye,
											p_Input, p_Output, dev_rmat, p_Samples);
	hipFree( dev_rmat );
	hipFree( dev_fov );
	hipFree( dev_fisheye );
}
